#include "hip/hip_runtime.h"
/*
 * Keccak 256
 *
 */

 extern "C"
 {
 #include "sph/sph_shavite.h"
 #include "sph/sph_simd.h"
 #include "sph/sph_keccak.h"
 }
 #include "miner.h"
 
 
 #include "cuda_helper.h"
 
 extern void keccak256_cpu_init(int thr_id, uint32_t threads);
 extern void keccak256_setBlock_ZP(int thr_id, void *pdata,const void *ptarget);
 extern void keccak256_cpu_hash_ZP(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *h_nounce);
 
 // CPU Hash
 void keccak256_zenprotocol_hash(void *state, const void *input)
 {
	 sph_keccak_context ctx_keccak;
 
	 uint32_t hash[16];
 
	 sph_keccak256_init(&ctx_keccak);
	 sph_keccak256 (&ctx_keccak, input, 100);
	 sph_keccak256_close(&ctx_keccak, (void*) hash);
 
	 memcpy(state, hash, 32);
 }

 extern int scanhash_keccak256_zenprotocol(int thr_id, uint32_t *pdata,
	 uint32_t *ptarget, uint32_t max_nonce,
	 uint32_t *hashes_done)
 {
	 static THREAD uint32_t *h_nounce = nullptr;
 
	 const uint32_t first_nonce = pdata[24];
	 uint32_t intensity = (device_sm[device_map[thr_id]] > 500) ? 1 << 28 : 1 << 27;;
	 uint32_t throughputmax = device_intensity(device_map[thr_id], __func__, intensity); // 256*4096
	 uint32_t throughput = min(throughputmax, max_nonce - first_nonce) & 0xfffffc00;
 
 
	 if (opt_benchmark)
		 ptarget[0] = 0x0002;
 
	 static THREAD volatile bool init = false;
	 if(!init)
	 {
		 if(throughputmax == intensity)
			 applog(LOG_INFO, "GPU #%d: using default intensity %.3f", device_map[thr_id], throughput2intensity(throughputmax));

		 CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		 CUDA_SAFE_CALL(hipDeviceReset());
		 CUDA_SAFE_CALL(hipSetDeviceFlags(cudaschedule));
		 CUDA_SAFE_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
		 CUDA_SAFE_CALL(hipStreamCreate(&gpustream[thr_id]));
		 CUDA_SAFE_CALL(hipHostMalloc(&h_nounce, 2 * sizeof(uint32_t)));
		 keccak256_cpu_init(thr_id, (int)throughputmax);
		 mining_has_stopped[thr_id] = false;
		 init = true;
	 }
 	 
	 keccak256_setBlock_ZP(thr_id, (void*)pdata, ptarget);

	 do {
 
		 keccak256_cpu_hash_ZP(thr_id, (int) throughput, pdata[24], h_nounce);
		 if(stop_mining) {mining_has_stopped[thr_id] = true; hipStreamDestroy(gpustream[thr_id]); pthread_exit(nullptr);}
		 if(h_nounce[0] != UINT32_MAX)
		 {
			 uint32_t Htarg = ptarget[0];
			 uint32_t vhash64[8]={0};
			 if(opt_verify){
				 pdata[24] = h_nounce[0];
                 keccak256_zenprotocol_hash(&vhash64[0], &pdata[0]);
			 }

			 if (swab32(vhash64[0]) <= Htarg && fulltest(vhash64, ptarget))
			 {
				 if (opt_debug)
				 {
					 char *hash_str;
					 hash_str = bin2hex((unsigned char *)vhash64, 32);
					 applog(LOG_DEBUG, "Hash: %s", hash_str);
					 free(hash_str);
				 }

				 int res = 1;
				 // check if there was some other ones...
				 *hashes_done = pdata[24] - first_nonce + throughput;
				 if (h_nounce[1] != 0xffffffff)
				 {
					 if(opt_verify){
						 pdata[24] = h_nounce[1];
						keccak256_zenprotocol_hash(vhash64, pdata);
						if (opt_debug)
						{
							char *hash_str;
							hash_str = bin2hex((unsigned char *)vhash64, 32);
							applog(LOG_DEBUG, "Hash2: %s", hash_str);
							free(hash_str);
						}
					 }
					 if (vhash64[0] <= Htarg && fulltest(vhash64, ptarget))
					 {
						 pdata[26] = h_nounce[1];
						 res++;
						 if (opt_benchmark)
							 applog(LOG_INFO, "GPU #%d Found second nonce %08x", device_map[thr_id], h_nounce[1]);
					 }
				 }
				 pdata[24] = h_nounce[0];
				 if (opt_benchmark)
					 applog(LOG_INFO, "GPU #%d Found nonce %08x", device_map[thr_id], h_nounce[0]);
				 return res;
			 }
		 }
 
		 pdata[24] += throughput;
		 //CUDA_SAFE_CALL(hipGetLastError());

	 } while (!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[24]) + (uint64_t)throughput)));
	 *hashes_done = pdata[24] - first_nonce ;
	 return 0;
 }
 